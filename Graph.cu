#include "hip/hip_runtime.h"
/*
 * Title: CS6023, GPU Programming, Jan-May 2023, Assignment-3
 * Description: Activation Game 
 */

#include <cstdio>        // Added for printf() function 
#include <sys/time.h>    // Added to get time of day
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <fstream>
#include "graph.hpp"
 
using namespace std;

ofstream outfile; // The handle for printing the output

/******************************Write your kerenels here ************************************/

// Calculating maximum node for a level
__global__ void find_Level(int *d_csrList, int *d_offset, int *d_max_Index, int s_Level, int e_Level){
    
    // c_node is present node
    int c_node = blockIdx.x * blockDim.x + threadIdx.x + s_Level;  
    
    if(c_node <= e_Level){
        // checking max node 
        if(d_csrList[d_offset[c_node+1]-1] > d_max_Index[0]){
            atomicMax(&d_max_Index[0], d_csrList[d_offset[c_node+1]-1]);
        }
    }

}
       
// Calculating indegree of each node in a level      
__global__ void in_Degree(int *d_csrList, int *d_offset, int *d_aid, int s_Level, int e_Level, int *d_active_node){
    
    // c_node is present node
    int c_node = blockIdx.x * blockDim.x + threadIdx.x + s_Level;

    if(c_node <= e_Level){
        int len = d_offset[c_node+1] - d_offset[c_node];
        // if edge from node is active increment indegree of present node
        if(d_active_node[c_node]){
            for(int i=0; i<len; i++){
                atomicAdd(&d_aid[d_csrList[d_offset[c_node]+i]], 1);
            }
        }
    }

}
    
// Calculating node is active or not in each level    
__global__ void active_Degree(int *d_aid, int s_Level, int e_Level, int *d_active_node, int *d_apr){
    
    // c_node is present node
    int c_node = blockIdx.x * blockDim.x + threadIdx.x + s_Level;

    if(c_node <= e_Level){
        // comparing apr and d_aid to check node is active or not
        if(d_aid[c_node] >= d_apr[c_node]){
            d_active_node[c_node] = 1;
        }    
    }

}   

// Checking for deactivation rule 2 at each level
__global__ void d_active_Degree(int s_Level, int e_Level, int *d_active_node, int *d_apr, int V){
    
    // c_node is present node
    int c_node = blockIdx.x * blockDim.x + threadIdx.x + s_Level;

    if(c_node <= e_Level){
        // Checking left and right node is active or not
        if(c_node > s_Level && c_node < e_Level){
            // If left and right node are inactive present node is set to inactive
            if(d_active_node[c_node-1]==0 && d_active_node[c_node+1]==0){
                d_active_node[c_node] = 0;
            }  
        } 
    } 

}   

// Calculating active nodes at each level
__global__ void active_nodes_Level(int s_Level, int e_Level, int *d_active_node, int *d_cnt){
    
    // c_node is present node
    int c_node = blockIdx.x * blockDim.x + threadIdx.x + s_Level;

    if(c_node <= e_Level){
          if(d_active_node[c_node]){
              atomicAdd(&d_cnt[0], 1);   // Adds 1 to d_cnt for each active node
          }
    }
}
    
/**************************************END*************************************************/



//Function to write result in output file
void printResult(int *arr, int V,  char* filename){
    outfile.open(filename);
    for(long int i = 0; i < V; i++){
        outfile<<arr[i]<<" ";   
    }
    outfile.close();
}

/**
 * Timing functions taken from the matrix multiplication source code
 * rtclock - Returns the time of the day 
 * printtime - Prints the time taken for computation 
 **/
double rtclock(){
    struct timezone Tzp;
    struct timeval Tp;
    int stat;
    stat = gettimeofday(&Tp, &Tzp);
    if (stat != 0) printf("Error return from gettimeofday: %d", stat);
    return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime){
    printf("%s%3f seconds\n", str, endtime - starttime);
}

int main(int argc,char **argv){
    // Variable declarations
    int V ; // Number of vertices in the graph
    int E; // Number of edges in the graph
    int L; // number of levels in the graph

    //Reading input graph
    char *inputFilePath = argv[1];
    graph g(inputFilePath);

    //Parsing the graph to create csr list
    g.parseGraph();

    //Reading graph info 
    V = g.num_nodes();
    E = g.num_edges();
    L = g.get_level();


    //Variable for CSR format on host
    int *h_offset; // for csr offset
    int *h_csrList; // for csr
    int *h_apr; // active point requirement

    //reading csr
    h_offset = g.get_offset();
    h_csrList = g.get_csr();   
    h_apr = g.get_aprArray();
    
    // Variables for CSR on device
    int *d_offset;
    int *d_csrList;
    int *d_apr; //activation point requirement array
    int *d_aid; // acive in-degree array
    //Allocating memory on device 
    hipMalloc(&d_offset, (V+1)*sizeof(int));
    hipMalloc(&d_csrList, E*sizeof(int)); 
    hipMalloc(&d_apr, V*sizeof(int)); 
    hipMalloc(&d_aid, V*sizeof(int));

    //copy the csr offset, csrlist and apr array to device
    hipMemcpy(d_offset, h_offset, (V+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_csrList, h_csrList, E*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_apr, h_apr, V*sizeof(int), hipMemcpyHostToDevice);

    // variable for result, storing number of active vertices at each level, on host
    int *h_activeVertex;
    h_activeVertex = (int*)malloc(L*sizeof(int));
    // setting initially all to zero
    memset(h_activeVertex, 0, L*sizeof(int));

    // variable for result, storing number of active vertices at each level, on device
    // int *d_activeVertex;
	  // hipMalloc(&d_activeVertex, L*sizeof(int));

    /***Important***/

    // Initialize d_aid array to zero for each vertex
    // Make sure to use comments

    /***END***/
    double starttime = rtclock(); 

    /*********************************CODE AREA*****************************************/

    //Variables on host  
    int *h_max_Index;     // max node at a level
    int *level_Start;     // array stores start index of each level
    int *level_End;       // array stores end index of each level
    int *h_active_node;   // array stores active or not of each node
    int *h_cnt;           // Number of active nodes at each level 

    //Allocating memory on host
    h_cnt = (int*)malloc(sizeof(int));
    h_max_Index = (int*)malloc(sizeof(int));
    level_Start = (int*)malloc(L*sizeof(int));
    level_End = (int*)malloc(L*sizeof(int));
    h_active_node = (int*)malloc(V*sizeof(int));

    memset(h_active_node, 0, V*sizeof(int)); // setting initially all to zero

    //Variables on devide
    int *d_max_Index;
    int *d_active_node;
    int *d_cnt;

    //Allocating memory on device and 
    hipMalloc(&d_cnt, sizeof(int));
    hipMalloc(&d_max_Index, sizeof(int));  
    hipMalloc(&d_active_node, V*sizeof(int));
    
    // setting initially all to zero
    hipMemset(d_max_Index, 0, sizeof(int));
    hipMemset(d_aid, 0, V*sizeof(int));
    hipMemset(d_active_node, 0, V*sizeof(int));

    int zero_Level_count = 0; // for number of nodes in level zero
    int threads = 1024;       // Number of threads in launching the kernel
    int block;                // Number of blocks in launching the kernel
    
    // Finding number of nodes at level zero
    for(int i=0; i<V; i++){
        if(h_apr[i] == 0){
            zero_Level_count++;
        }
        else{
            break;
        }
    }

    // setting start and end node of level zero
    level_Start[0] = 0;
    level_End[0] = zero_Level_count-1;
    
    // setting active node of level zero to 1
    for(int i=0; i<=level_End[0]; i++){
        h_active_node[i] = 1;
    }
    //copy the h_active_node array to device
    hipMemcpy(d_active_node, h_active_node, V*sizeof(int), hipMemcpyHostToDevice);

    // Finding start and end node of each level
    for(int i=0; i<L-1; i++){

        // Number of blocks for kernel launch
        block = ceil((float)(level_End[i]-level_Start[i]+1)/1024); 
        // Finding max node at level
        find_Level<<<block, threads>>>(d_csrList, d_offset, d_max_Index, level_Start[i], level_End[i]);
        //copy the d_max_Index array to host
        hipMemcpy(h_max_Index, d_max_Index, sizeof(int), hipMemcpyDeviceToHost);
        // setting start and end node of level
        level_Start[i+1] = level_End[i]+1;
        level_End[i+1] = h_max_Index[0];

    }

    // Finding node is active or not 
    for(int i=0; i<L-1; i++){

        block = ceil((float)(level_End[i]-level_Start[i]+1)/1024);   // Number of blocks for kernel launch
        // Kernel call finds indegree of nodes
        in_Degree<<<block, threads>>>(d_csrList, d_offset, d_aid, level_Start[i], level_End[i], d_active_node);
        hipDeviceSynchronize();

        // Number of blocks for kernel launch
        block = ceil((float)(level_End[i+1]-level_Start[i+1]+1)/1024);  
        // Kernel call finds active or not of nodes by comparing apr and d_indegre
        active_Degree<<<block, threads>>>(d_aid, level_Start[i+1], level_End[i+1], d_active_node, d_apr);
        hipDeviceSynchronize();
        
        //Kernel call checks for left and right active and sets 
        d_active_Degree<<<block, threads>>>(level_Start[i+1], level_End[i+1], d_active_node, d_apr, V);
        hipDeviceSynchronize();

    }   
    
    // Calculating active nodes in the level
    for(int i=0; i<L; i++){
        // setting initially to zero before each kernel call
        hipMemset(d_cnt, 0, sizeof(int)); 
        // Number of blocks for kernel launch
        block = ceil((float)(level_End[i]-level_Start[i]+1)/1024);
        // Kernel gives active nodes in the level
        active_nodes_Level<<<block, threads>>>(level_Start[i], level_End[i], d_active_node, d_cnt);
        //copy the d_cnt array to host
        hipMemcpy(h_cnt, d_cnt, sizeof(int), hipMemcpyDeviceToHost);
        // setting h_cnt in h_activeVertex
        h_activeVertex[i] = h_cnt[0];
    }
        

    /********************************END OF CODE AREA**********************************/
    double endtime = rtclock();  
    printtime("GPU Kernel time: ", starttime, endtime);  

    // --> Copy C from Device to Host
    char outFIle[30] = "./output.txt" ;
    printResult(h_activeVertex, L, outFIle);
    if(argc>2)
    {
        for(int i=0; i<L; i++)
        {
            printf("level = %d , active nodes = %d\n",i,h_activeVertex[i]);
        }
    }

    return 0;
}
